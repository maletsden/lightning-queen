#include "hip/hip_runtime.h"
#include "analyzer.cuh"

#include <iostream>
#include <vector>

#include <validator/validator.h>
#include <cuda_validator/cuda_validator.h>


__global__ void analyzer::analyzeGenome(
    const char *device_genome_buffer, RESULT_T *results_vector, std::size_t genome_size
) {
  const auto i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= genome_size) return;

  const std::uint8_t nucleotide = device_genome_buffer[i];

  constexpr std::uint8_t charACode = 'A';

  atomicAdd(results_vector + nucleotide - charACode, 1);
}


void analyzer::analyze(ThreadSafeQueue<std::string> &genomes_queue) {
  constexpr RESULT_T expected_genome_size = 100 * 1024 * 1024; // 100 MB
  constexpr int threadsPerBlock = 256;
  constexpr int blocksPerGrid = (expected_genome_size + threadsPerBlock - 1) / threadsPerBlock;

  hipDeviceProp_t prop{};
  cuda_validator::check_error(hipGetDeviceProperties(&prop, 0));
  std::cout << "Device: " << prop.name << std::endl;

  // Allocate genome buffer on device.
  char *device_genome_buffer;

  cuda_validator::check_error(hipMalloc(&device_genome_buffer, expected_genome_size)); // device

  // Allocate result vector on device.
  RESULT_T *results_vector;

  constexpr RESULT_T results_buffer_size = 20;
  constexpr RESULT_T results_buffer_size_in_bytes = results_buffer_size * sizeof(RESULT_T);

  cuda_validator::check_error(hipMalloc(&results_vector, results_buffer_size_in_bytes)); // device

  std::vector<RESULT_T> host_result_vector(results_buffer_size);

  // Events for timing.
  hipEvent_t startEvent, stopEvent;

  cuda_validator::check_error(hipEventCreate(&startEvent));
  cuda_validator::check_error(hipEventCreate(&stopEvent));


  while (true) {
    const auto file = genomes_queue.dequeue();

    // Catch poison pill.
    if (file.empty()) break;

    std::cout << "Transfer size (MB): " << file.size() / (1024 * 1024) << std::endl;

    cuda_validator::check_error(hipEventRecord(startEvent, nullptr));

    cuda_validator::check_error(hipMemset(results_vector, 0, results_buffer_size_in_bytes));

    cuda_validator::check_error(hipMemcpy(device_genome_buffer, file.data(), file.size(), hipMemcpyHostToDevice));

    // Invoke kernel.
    analyzeGenome<<<blocksPerGrid, threadsPerBlock>>>(device_genome_buffer, results_vector, file.size());

    cuda_validator::check_error(hipEventSynchronize(stopEvent));

    cuda_validator::check_error(
        hipMemcpy(
            host_result_vector.data(), results_vector, results_buffer_size_in_bytes, hipMemcpyDeviceToHost
        )
    );

    cuda_validator::check_error(hipEventRecord(stopEvent, nullptr));
    cuda_validator::check_error(hipEventSynchronize(stopEvent));

    // Check results.
    validator::validate_results(host_result_vector);

    float time;
    cuda_validator::check_error(hipEventElapsedTime(&time, startEvent, stopEvent));
    std::cout << "  Host to Device bandwidth (GB/s): " << static_cast<double>(file.size()) * 1e-6 / time << std::endl;
  }

  // Clean up events.
  cuda_validator::check_error(hipEventDestroy(startEvent));
  cuda_validator::check_error(hipEventDestroy(stopEvent));

  hipFree(device_genome_buffer);
}