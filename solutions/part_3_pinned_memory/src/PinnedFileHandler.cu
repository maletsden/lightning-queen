#include "PinnedFileHandler.cuh"

#include <hip/hip_runtime.h>
#include <cuda_validator/cuda_validator.h>

PinnedFileHandler::PinnedFileHandler(size_t filesize) : size(filesize) {
  cuda_validator::check_error(hipHostAlloc(&data, filesize, hipHostMallocDefault));
}

PinnedFileHandler::~PinnedFileHandler() {
  if (data) hipHostFree(data);
}
