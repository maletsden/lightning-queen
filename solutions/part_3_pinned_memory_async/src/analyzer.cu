#include "hip/hip_runtime.h"
#include "analyzer.cuh"

#include <iostream>
#include <vector>
#include <array>
#include <cassert>

#include <validator/validator.h>
#include <cuda_validator/cuda_validator.h>
#include <cuda_stopwatch/CudaStopwatch.cuh>


__global__ void analyzer::analyze_genome(
    const char *device_genome_buffer, RESULT_T *results_vector, std::size_t genome_size
) {
  const auto i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= genome_size) return;

  const std::uint8_t nucleotide = device_genome_buffer[i];

  constexpr std::uint8_t charACode = 'A';

  atomicAdd(results_vector + threadIdx.x * CACHE_LINE_SIZE + nucleotide - charACode, 1);
}


void analyzer::analyze(QUEUE_T &genomes_queue) {

  constexpr RESULT_T expected_genome_size = 100 * 1024 * 1024; // 100 MB
  constexpr int threadsPerBlock = 256;
  constexpr int blocksPerGrid = (expected_genome_size + threadsPerBlock - 1) / threadsPerBlock;
  hipDeviceProp_t prop{};
  cuda_validator::check_error(hipGetDeviceProperties(&prop, 0));

  std::cout << "Device: " << prop.name << std::endl;
  std::cout << "asyncEngineCount: " << prop.asyncEngineCount << std::endl;

  // Allocate genome buffer on device.
  char *device_genome_buffer;
  cuda_validator::check_error(hipMalloc(&device_genome_buffer, expected_genome_size)); // device

  // Allocate result vector on device.
  RESULT_T *results_vector;
  constexpr RESULT_T results_buffer_size = CACHE_LINE_SIZE * threadsPerBlock * 20;
  constexpr RESULT_T results_buffer_size_in_bytes = results_buffer_size * sizeof(RESULT_T);

  cuda_validator::check_error(hipMalloc(&results_vector, results_buffer_size_in_bytes)); // device

  std::vector<RESULT_T> host_result_vector(results_buffer_size);
  std::vector<RESULT_T> host_result_vector_total(results_buffer_size / threadsPerBlock);

  CudaStopwatch cuda_stopwatch;

  constexpr auto n_streams = 4;

  hipStream_t streams[n_streams];
  for (auto &stream: streams) {
    cuda_validator::check_error(hipStreamCreate(&stream));
  }
  constexpr auto stream_genome_size = expected_genome_size / n_streams;
  constexpr auto stream_blocks_per_grid = blocksPerGrid / n_streams;


  while (true) {
    auto file_handler = genomes_queue.dequeue();

    // Catch poison pill.
    if (file_handler->is_empty()) break;

    const auto filesize = file_handler->get_size();

    cuda_stopwatch.start(filesize);

    cuda_validator::check_error(hipMemset(results_vector, 0, results_buffer_size_in_bytes));

    for (auto i = 0; i < n_streams; i++) {
      const auto offset = i * stream_genome_size;
      cuda_validator::check_error(
          hipMemcpyAsync(device_genome_buffer + offset, file_handler->get_data() + offset, stream_genome_size,
                          hipMemcpyHostToDevice, streams[i]));
      analyze_genome<<<stream_blocks_per_grid, threadsPerBlock, 0, streams[i]>>>(device_genome_buffer + offset,
                                                                                 results_vector,
                                                                                 stream_genome_size);
    }

    cuda_validator::check_error(
        hipMemcpy(host_result_vector.data(), results_vector, results_buffer_size_in_bytes, hipMemcpyDeviceToHost));

    std::fill(host_result_vector_total.begin(), host_result_vector_total.end(), 0);

    for (int i = 0; i < threadsPerBlock; ++i) {
      host_result_vector_total['A' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'A' - 'A'];
      host_result_vector_total['C' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'C' - 'A'];
      host_result_vector_total['G' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'G' - 'A'];
      host_result_vector_total['N' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'N' - 'A'];
      host_result_vector_total['T' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'T' - 'A'];
    }

    cuda_stopwatch.stop();

    // Check results.
    validator::validate_results(host_result_vector_total);
  }

  for (auto &stream: streams) {
    cuda_validator::check_error(hipStreamDestroy(stream));
  }

  hipFree(device_genome_buffer);
}
