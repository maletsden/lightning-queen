#include "PinnedMemoryHandler.cuh"

#include <hip/hip_runtime.h>
#include <cuda_validator/cuda_validator.h>

PinnedMemoryHandler::PinnedMemoryHandler(size_t filesize) : size(filesize) {
  cuda_validator::check_error(hipHostAlloc(&data, filesize, hipHostMallocDefault));
}

PinnedMemoryHandler::~PinnedMemoryHandler() {
  if (data) hipHostFree(data);
}
