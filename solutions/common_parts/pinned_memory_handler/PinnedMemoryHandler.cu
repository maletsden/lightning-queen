#include "PinnedMemoryHandler.cuh"

#include <hip/hip_runtime.h>
#include <cuda_validator/cuda_validator.h>

PinnedMemoryHandler::PinnedMemoryHandler(size_t filesize, unsigned int flags) : size(filesize) {
  cuda_validator::check_error(hipHostAlloc(&data, filesize, flags));
}

PinnedMemoryHandler::~PinnedMemoryHandler() {
  if (data) hipHostFree(data);
}
