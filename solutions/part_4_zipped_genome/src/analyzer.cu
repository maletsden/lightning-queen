#include "hip/hip_runtime.h"
#include "analyzer.cuh"

#include <iostream>
#include <vector>
#include <cassert>
#include <string>

#include <validator/validator.h>
#include <cuda_validator/cuda_validator.h>
#include <cuda_stopwatch/CudaStopwatch.cuh>


__global__ void analyzer::analyze_genome(
    const char *device_genome_buffer, RESULT_T *results_vector, std::size_t zipped_genome_size
) {
  const auto i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= zipped_genome_size) return;

  const char zipped_nucleotides = device_genome_buffer[i];

  constexpr char decodeChar[4] = {
      'A', 'C', 'G', 'T'
  };
  constexpr auto first_2_bits = 0b11000000;
  constexpr auto second_2_bits = 0b00110000;
  constexpr auto third_2_bits = 0b00001100;
  constexpr auto fourth_2_bits = 0b00000011;
  const auto thread_result_offset = results_vector + threadIdx.x * CACHE_LINE_SIZE;

  const std::uint8_t N_num = zipped_nucleotides & first_2_bits;

  switch (N_num) {
    case 0b00000000:
      atomicAdd(thread_result_offset + decodeChar[(zipped_nucleotides & second_2_bits) >> 4] - 'A', 1);
      atomicAdd(thread_result_offset + decodeChar[(zipped_nucleotides & third_2_bits) >> 2] - 'A', 1);
      atomicAdd(thread_result_offset + decodeChar[zipped_nucleotides & fourth_2_bits] - 'A', 1);
      break;
    case 0b01000000: {
      atomicAdd(thread_result_offset + 'N' - 'A', 1);

      // decode 2 "not N" chars
      atomicAdd(thread_result_offset + decodeChar[(zipped_nucleotides & third_2_bits) >> 2] - 'A', 1);
      atomicAdd(thread_result_offset + decodeChar[zipped_nucleotides & fourth_2_bits] - 'A', 1);
      break;
    }
    case 0b10000000: {
      atomicAdd(thread_result_offset + 'N' - 'A', 2);
      // decode "not N" char
      atomicAdd(thread_result_offset + decodeChar[(zipped_nucleotides & third_2_bits) >> 2] - 'A', 1);
      break;
    }
    default:
      // in other case (N_num == 3) we can just add them
      atomicAdd(thread_result_offset + 'N' - 'A', 3);
      break;
  }
}

std::vector<char> analyze_last_zipped_char(const char zipped_nucleotides) {
  constexpr char decodeChar[4] = {
      'A', 'C', 'G', 'T'
  };
  constexpr auto first_2_bits = 0b11000000;
  constexpr auto second_2_bits = 0b00110000;
  constexpr auto third_2_bits = 0b00001100;
  constexpr auto fourth_2_bits = 0b00000011;

  const std::uint8_t N_num = zipped_nucleotides & first_2_bits;

  switch (N_num) {
    case 0b00000000:
      return {
          decodeChar[(zipped_nucleotides & second_2_bits) >> 4],
          decodeChar[(zipped_nucleotides & third_2_bits) >> 2],
          decodeChar[(zipped_nucleotides & fourth_2_bits)],
      };
    case 0b01000000: {
      std::vector<char> decoded = {decodeChar[(zipped_nucleotides & third_2_bits) >> 2],
                                   decodeChar[zipped_nucleotides & fourth_2_bits]};
      const std::uint8_t N_index = (zipped_nucleotides & second_2_bits) >> 4;
      decoded.insert(decoded.begin() + N_index, 'N');
      return decoded;
    }
    case 0b10000000: {
      std::vector<char> decoded = {'N', 'N'};
      const std::uint8_t non_N_index = (zipped_nucleotides & second_2_bits) >> 4;
      decoded.insert(decoded.begin() + non_N_index, decodeChar[(zipped_nucleotides & third_2_bits) >> 2]);
      return decoded;
    }
    default:
      return {'N', 'N', 'N'};
  }
}

void analyzer::analyze(QUEUE_T &zipped_genomes_queue) {

  constexpr int threadsPerBlock = 256;
  constexpr int blocksPerGrid = (EXPECTED_GENOME_SIZE + threadsPerBlock - 1) / threadsPerBlock;
  hipDeviceProp_t prop{};
  cuda_validator::check_error(hipGetDeviceProperties(&prop, 0));

  std::cout << "Device: " << prop.name << std::endl;

  if (!prop.canMapHostMemory) {
    std::cerr << "Mapped Host memory is not supported for this device";
    exit(EXIT_FAILURE);
  }
  hipSetDeviceFlags(hipDeviceMapHost);

  // Allocate result vector on device.
  RESULT_T *results_vector;
  constexpr RESULT_T results_buffer_size = CACHE_LINE_SIZE * threadsPerBlock * 20;
  constexpr RESULT_T results_buffer_size_in_bytes = results_buffer_size * sizeof(RESULT_T);

  cuda_validator::check_error(hipMalloc(&results_vector, results_buffer_size_in_bytes)); // device

  std::vector<RESULT_T> host_result_vector(results_buffer_size);
  std::vector<RESULT_T> host_result_vector_total(results_buffer_size / threadsPerBlock);

  CudaStopwatch cuda_stopwatch;

  while (true) {
    auto zipped_genome_handler = zipped_genomes_queue.dequeue();

    // Catch poison pill.
    if (zipped_genome_handler.is_empty()) break;

    const auto filesize = zipped_genome_handler.real_size;
    std::cout << "Transfer m_size (MB): " << filesize / (1024 * 1024) << std::endl;

    cuda_stopwatch.start(filesize);

    cuda_validator::check_error(hipMemset(results_vector, 0, results_buffer_size_in_bytes));

    char *device_genome_buffer;
    cuda_validator::check_error(
        hipHostGetDevicePointer(&device_genome_buffer, zipped_genome_handler.container->get_data(), 0));

    // Invoke kernel.
    analyze_genome<<<blocksPerGrid, threadsPerBlock>>>(device_genome_buffer, results_vector,
                                                       zipped_genome_handler.container->get_size() - 1);


    cuda_validator::check_error(
        hipMemcpy(host_result_vector.data(), results_vector, results_buffer_size_in_bytes, hipMemcpyDeviceToHost));

    std::fill(host_result_vector_total.begin(), host_result_vector_total.end(), 0);

    for (int i = 0; i < threadsPerBlock; ++i) {
      host_result_vector_total['A' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'A' - 'A'];
      host_result_vector_total['C' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'C' - 'A'];
      host_result_vector_total['G' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'G' - 'A'];
      host_result_vector_total['N' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'N' - 'A'];
      host_result_vector_total['T' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'T' - 'A'];
    }

    const auto last_decode_char = analyze_last_zipped_char(
        zipped_genome_handler.container->get_data()[zipped_genome_handler.container->get_size() - 1]);
    for (int i = 0; i < filesize % 3; ++i) {
      ++host_result_vector_total[last_decode_char[i] - 'A'];
    }

    cuda_stopwatch.stop();

    // Check results.
    validator::validate_results(host_result_vector_total);
  }

  hipFree(results_vector);
}
