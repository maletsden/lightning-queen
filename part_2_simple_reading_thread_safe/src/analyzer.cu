#include "hip/hip_runtime.h"
#include "../inc/analyzer.cuh"


__global__ void analyzeGenome(
    const char *device_genome_buffer, std::uint32_t *results_vector, std::size_t genome_size
) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= genome_size) return;

  const std::uint8_t nucleotide = device_genome_buffer[i];

  constexpr std::uint8_t charACode = 'A';

  atomicAdd(results_vector + threadIdx.x * CACHE_LINE_SIZE + nucleotide - charACode, 1);
}


void run_analyzer(ThreadSafeQueue<std::string>& genomes_queue) {

  constexpr std::uint32_t expected_genome_size = 100 * 1024 * 1024; // 100 MB
  constexpr int threadsPerBlock = 256;
  constexpr int blocksPerGrid = (expected_genome_size + threadsPerBlock - 1) / threadsPerBlock;
  hipDeviceProp_t prop{};
  checkCuda(hipGetDeviceProperties(&prop, 0));

  std::cout << "Device: " << prop.name << std::endl;

  // Allocate genome buffer on device.
  char *device_genome_buffer;
  checkCuda(hipMalloc(&device_genome_buffer, expected_genome_size)); // device

  // Allocate result vector on device.
  std::uint32_t *results_vector;
  constexpr std::uint32_t results_buffer_size = CACHE_LINE_SIZE * threadsPerBlock;
  constexpr std::uint32_t results_buffer_size_in_bytes = results_buffer_size * sizeof(std::uint32_t);

  checkCuda(hipMalloc(&results_vector, results_buffer_size_in_bytes)); // device

  std::vector<std::uint32_t> host_result_vector(results_buffer_size);
  std::vector<std::uint32_t> host_result_vector_total(results_buffer_size / threadsPerBlock);

  // Events for timing.
  hipEvent_t startEvent, stopEvent;

  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));


  while (true) {
    auto file = genomes_queue.dequeue();

    // Catch poison pill.
    if (file.empty()) break;

    printf("Transfer size (MB): %lu\n", file.size() / (1024 * 1024));

    checkCuda(hipEventRecord(startEvent, nullptr));

    checkCuda(hipMemset(results_vector, 0, results_buffer_size_in_bytes));

    checkCuda(hipMemcpy(device_genome_buffer, file.data(), file.size(), hipMemcpyHostToDevice));

    // Invoke kernel.
    analyzeGenome<<<blocksPerGrid, threadsPerBlock>>>(device_genome_buffer, results_vector, file.size());

    checkCuda(hipEventSynchronize(stopEvent));

    checkCuda(
        hipMemcpy(host_result_vector.data(), results_vector, results_buffer_size_in_bytes, hipMemcpyDeviceToHost));

    std::fill(host_result_vector_total.begin(), host_result_vector_total.end(), 0);

    for (int i = 0; i < threadsPerBlock; ++i) {
      host_result_vector_total['A' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'A' - 'A'];
      host_result_vector_total['C' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'A' - 'A'];
      host_result_vector_total['G' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'A' - 'A'];
      host_result_vector_total['N' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'A' - 'A'];
      host_result_vector_total['T' - 'A'] += host_result_vector[i * CACHE_LINE_SIZE + 'A' - 'A'];
    }

    checkCuda(hipEventRecord(stopEvent, nullptr));
    checkCuda(hipEventSynchronize(stopEvent));

    // Check results.
    check_results(host_result_vector_total);

    float time;
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
    printf("  Host to Device bandwidth (GB/s): %f\n", static_cast<double>(file.size()) * 1e-6 / time);

  }

  // Clean up events.
  checkCuda(hipEventDestroy(startEvent));
  checkCuda(hipEventDestroy(stopEvent));

  hipFree(device_genome_buffer);

}